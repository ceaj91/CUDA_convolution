#include <stdio.h>
#include <hip/hip_runtime.h>
#define INPUT_SIZE 1024
#define KERNEL_SIZE 9
#define OUTPUT_SIZE 1024
#define TILE_SIZE 256

__constant__ float d_kernel[KERNEL_SIZE];

__global__ void conv_1D_tiled_halo_elements(float *in_data, float *out_data, int kernel_width, int data_width)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float in_data_ds[TILE_SIZE + KERNEL_SIZE - 1];

    int n = kernel_width/2;

    int halo_index_left = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
    if(threadIdx.x >= blockDim.x - n)
    {
        in_data_ds[threadIdx.x - (blockDim.x - n)] = (halo_index_left < 0) ? 0 : in_data[halo_index_left];
    }


    in_data_ds[n + threadIdx.x] = in_data[blockIdx.x * blockDim.x + threadIdx.x];


    int halo_index_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
    if(threadIdx.x < n)
    {
        in_data_ds[n + blockDim.x + threadIdx.x] = (halo_index_right >= data_width) ? 0 : in_data[halo_index_right];
    }

    __syncthreads();

    float Pvalue = 0;

    for(int j=0; j<kernel_width; j++)
    {
    	Pvalue += in_data_ds[threadIdx.x + j] * d_kernel[j];
    }

    out_data[i] = Pvalue;
}




int main()
{

	float *input_array  = (float*)malloc(INPUT_SIZE*sizeof(float));
	float *kernel       = (float*)malloc(KERNEL_SIZE*sizeof(float));
	float *output_array = (float*)malloc(OUTPUT_SIZE*sizeof(float));

	float *d_input_array, *d_output_array;
	hipMalloc(&d_input_array, INPUT_SIZE*sizeof(float));
    //cudaMalloc(&d_kernel, KERNEL_SIZE*sizeof(float));
    hipMalloc(&d_output_array, OUTPUT_SIZE*sizeof(float));

    for(int i=0; i<INPUT_SIZE; i++)
    {
        input_array[i] = (float)(rand()%10);
    }

    for(int i=0; i<KERNEL_SIZE; i++)
    {
        kernel[i] = (float)(rand()%5);
    }

    hipMemcpy(d_input_array, input_array, INPUT_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), kernel, KERNEL_SIZE*sizeof(float));


	int thr_per_blk = TILE_SIZE;
	int blk_in_grid = ceil( float(OUTPUT_SIZE) / thr_per_blk );
	conv_1D_tiled_halo_elements<<< blk_in_grid, thr_per_blk >>>(d_input_array, d_output_array, KERNEL_SIZE,INPUT_SIZE);

	hipMemcpy(output_array, d_output_array, OUTPUT_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0; i<INPUT_SIZE; i++)
    {
    	
    	printf( "in[%d] = %lf\n", i, input_array[i]);
        
    }
    for(int i=0; i<KERNEL_SIZE; i++)
    {
    	
    	printf( "kernel[%d] = %lf\n", i, kernel[i]);
        
    }
	for(int i=0; i<OUTPUT_SIZE; i++)
    {
    	printf( "out[%d] = %lf\n", i, output_array[i]);
       
    }

    

    free(input_array);
    //free(kernel);
    free(output_array);

    hipFree(d_input_array);
    hipFree(d_kernel);
    hipFree(d_output_array);

    printf("\n---------------------------\n");
    printf("__SUCCESS__\n");
    printf("---------------------------\n");
    printf("N                 = %d\n", INPUT_SIZE);
    printf("Threads Per Block = %d\n", thr_per_blk);
    printf("Blocks In Grid    = %d\n", blk_in_grid);
    printf("---------------------------\n\n");
}
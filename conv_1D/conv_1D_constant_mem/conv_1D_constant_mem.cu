#include <stdio.h>
#include <hip/hip_runtime.h>
#define INPUT_SIZE 1024
#define KERNEL_SIZE 9
#define OUTPUT_SIZE 1024


__constant__ float d_kernel[KERNEL_SIZE];


__global__ void conv_1D_basic( float *in_data, float *out_data, int kernel_width, int data_width)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float Pvalue = 0;
    int N_start_point = i - (kernel_width/2);

    for(int j=0; j<kernel_width; j++)
    {
    	if(N_start_point + j >= 0 && N_start_point + j < data_width)
    		Pvalue += in_data[N_start_point + j] * d_kernel[j];
    }
    out_data[i] = Pvalue;
}




int main()
{

	float *input_array  = (float*)malloc(INPUT_SIZE*sizeof(float));
	float *kernel       = (float*)malloc(KERNEL_SIZE*sizeof(float));
	float *output_array = (float*)malloc(OUTPUT_SIZE*sizeof(float));

	float *d_input_array, *d_output_array;
	hipMalloc(&d_input_array, INPUT_SIZE*sizeof(float));
	//cudaMalloc(&d_kernel, KERNEL_SIZE*sizeof(float));
	hipMalloc(&d_output_array, OUTPUT_SIZE*sizeof(float));

	for(int i=0; i<INPUT_SIZE; i++)
	{
		input_array[i] = (float)(rand()%10);
	}

	for(int i=0; i<KERNEL_SIZE; i++)
	{
		kernel[i] = (float)(rand()%5);
	}

	hipMemcpy(d_input_array, input_array, INPUT_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), kernel, KERNEL_SIZE*sizeof(float));


	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(OUTPUT_SIZE) / thr_per_blk );
	conv_1D_basic<<< blk_in_grid, thr_per_blk >>>(d_input_array, d_output_array, KERNEL_SIZE,INPUT_SIZE);

	hipMemcpy(output_array, d_output_array, OUTPUT_SIZE*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<INPUT_SIZE; i++)
	{

		printf( "in[%d] = %lf\n", i, input_array[i]);

	}
	for(int i=0; i<KERNEL_SIZE; i++)
	{
		printf( "kernel[%d] = %lf\n", i, kernel[i]);
	}
	
	for(int i=0; i<OUTPUT_SIZE; i++)
	{
		printf( "out[%d] = %lf\n", i, output_array[i]);
	}

    

	free(input_array);
	//free(kernel);
	free(output_array);

	hipFree(d_input_array);
	hipFree(d_kernel);
	hipFree(d_output_array);

	printf("\n---------------------------\n");
	printf("__SUCCESS__\n");
	printf("---------------------------\n");
	printf("N                 = %d\n", INPUT_SIZE);
	printf("Threads Per Block = %d\n", thr_per_blk);
	printf("Blocks In Grid    = %d\n", blk_in_grid);
	printf("---------------------------\n\n");
}
